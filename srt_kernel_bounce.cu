#include "hip/hip_runtime.h"

__global__ void bounce(float* to, float* from, rrnode *dev_bounce, 
	unsigned char *ncls, int revcount)
{
	int i, j, k, x, y, z;
	int n= get_global_id(0);
	if(n>=revcount)
		return; 
	i=dev_bounce[n].i;
	j=dev_bounce[n].j;
	k=dev_bounce[n].k;
	
	for(int l=0;l<DIRECTIONS;++l){
		
			x = i - ci[l].x;
			y = j - ci[l].y;
			z = k - ci[l].z; 
			if( dev_bounce[n].del[l]>-1 && ncls[cstore(i,j,k)] == BOUNDARY  && ncls[cstore(x,y,z)] == FFLOW ){
					float temp = from[store(x,y,z,l)];
					if(temp > 0){
							temp=-temp;						
					}
					to[store(x,y,z,l)]=temp;
				}		
			//	to[store(i,j,k,m)] = curr_value - sub ;
				
			
		
	}
		
		
	
//printf("I am in bounce");
return;
}

