#include "hip/hip_runtime.h"
__global__ void cascade(float* to, float* from, unsigned char *ncls)
{

	int i, j, k, x, y, z;
	i = get_global_id(0);
	j = get_global_id(1);
	k = get_global_id(2);

	if(ncls[cstore(i,j,k)] == FREE || ncls[cstore(i,j,k)] == FFLOW){
		struct rvector *uptr;
		double rhoptr;
		get_rho_u(from, i, j, k, rhoptr, uptr);
		for(int l=0;l<DIRECTIONS;++l){
			{
				float rho;
				struct rvector u;
				float equi=get_equilibrium(rho,u, l);
				
				float curr_value= from[store(x,y,z,m)];
				float sub =  omega* (curr_value - equi ); 
				to[store(i,j,k,m)] = curr_value - sub ;
				
			}
		
		}
		
		
	}


//printf("I am in cascade");
return;
}
