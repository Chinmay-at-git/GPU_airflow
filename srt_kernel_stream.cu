#include "hip/hip_runtime.h"
__global__ void stream(float* to, float* from, unsigned char *ncls)
{
	int i, j, k, x, y, z;
	i = get_global_id(0);
	j = get_global_id(1);
	k = get_global_id(2);

	if(ncls(cstore(i,j,k)) == FREE || ncls(cstore(i,j,k)) == FFLOW){
		x = i - ci[m].x;
		y = j - ci[m].y;
		z = k - ci[m].z;

		to[i,j,k,m] = from[x,y,z,m];
	}
//printf("I am in stream");
return;
}